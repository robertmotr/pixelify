#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "filters.h"
#include "filter_impl.h"
#include "gtest/gtest.h"
#include <string>

#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wunused-but-set-variable"

#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"

#pragma GCC diagnostic pop

const filter** filters = init_filters();

void print_to(const Pixel<3>& pixel, ::std::ostream* os) {
    // recall pixels have short4
    *os << "(" << pixel.data.x << ", " << pixel.data.y << ", " << pixel.data.z << ", " << pixel.data.w << ")";
}

void print_to(const Pixel<4>& pixel, ::std::ostream* os) {
    // recall pixels have short4
    *os << "(" << pixel.data.x << ", " << pixel.data.y << ", " << pixel.data.z << ", " << pixel.data.w << ")";
}

// Helper function to initialize an array of pixels
template<unsigned int channels>
void init_image(Pixel<channels> *d_image, int pixels) {
    for (int i = 0; i < pixels; ++i) {
        for (int channel = 0; channel < channels; ++channel) {
            d_image[i].set(channel, rand() % 256);
        }
    }
}

// CPU reduction function for verification
template <unsigned int channels>
Pixel<channels> cpu_image_reduction(const Pixel<channels> *image, int pixels, bool reduce_type) {
    Pixel<channels> result;
    for (int channel = 0; channel < channels; ++channel) {
        if (reduce_type == MAX_REDUCE) {
            result.set(channel, SHORT_MAX);
        } else {
            result.set(channel, SHORT_MIN);
        }
    }

    for (int i = 0; i < pixels; ++i) {
        for (int channel = 0; channel < channels; ++channel) {
            if (reduce_type == MAX_REDUCE) {
                result.set(channel, max(result.at(channel), image[i].at(channel)));
            } else {
                result.set(channel, min(result.at(channel), image[i].at(channel)));
            }
        }
    }
    return result;
}

TEST(KernelHelpers, find_index) {
    int index = find_index(0, 0, 3, 3);
    ASSERT_EQ(index, -1);
    index = find_index(3, 3, 0, 0);
    ASSERT_EQ(index, 0);
    index = find_index(3, 3, 1, 1);
    ASSERT_EQ(index, 4);
    index = find_index(3, 3, 2, 2);
    ASSERT_EQ(index, 8);
    index = find_index(3, 3, 2, 1);
    ASSERT_EQ(index, 7);
}

TEST(KernelHelpers, clamp_pixels) {
    Pixel<3> pixel = {-20, -230, 300};
    clamp_pixels<3>(&pixel, 0);
    ASSERT_EQ(pixel.data.x, 0);
    ASSERT_EQ(pixel.data.y, 0);
    ASSERT_EQ(pixel.data.z, 255);

    // create a list of pixels
    Pixel<3> pixels[3] = {{-20, -230, 300}, {0, 0, 0}, {255, 255, 255}};
    clamp_pixels<3>(pixels, 0);
    clamp_pixels<3>(pixels, 1);
    clamp_pixels<3>(pixels, 2);
    ASSERT_EQ(pixels[0].data.x, 0);
    ASSERT_EQ(pixels[0].data.y, 0);
    ASSERT_EQ(pixels[0].data.z, 255);
    ASSERT_EQ(pixels[1].data.x, 0);
    ASSERT_EQ(pixels[1].data.y, 0);
    ASSERT_EQ(pixels[1].data.z, 0);
    ASSERT_EQ(pixels[2].data.x, 255);
    ASSERT_EQ(pixels[2].data.y, 255);
    ASSERT_EQ(pixels[2].data.z, 255);
}

TEST(KernelHelpers, shift_colours) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    // blue values get multiplied by two
    Pixel<3> expected[16] = {
        {0, 0, 0}, {255, 255, 510}, {0, 0, 0}, {255, 255, 510},
        {0, 0, 0}, {255, 255, 510}, {0, 0, 0}, {255, 255, 510},
        {0, 0, 0}, {255, 255, 510}, {0, 0, 0}, {255, 255, 510},
        {0, 0, 0}, {255, 255, 510}, {0, 0, 0}, {255, 255, 510}
    };

    Pixel<3> *d_pixels;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);

    struct filter_args args;
    args.alpha_shift = 0;
    args.red_shift = 0;
    args.green_shift = 0;
    args.blue_shift = 100;

    shift_kernel<3><<<1, 1024>>>(d_pixels, 4, 4, args);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, 16 * sizeof(Pixel<3>), hipMemcpyDeviceToHost);

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(pixels[i], expected[i]) << "Mismatch at index " << i;
    }
    hipFree(d_pixels);
}

TEST(KernelHelpers, brightness_kernel_test) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> pixels_expected[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> *d_pixels;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);

    struct filter_args args;
    args.brightness = 0;

    brightness_kernel<3><<<1, 1024>>>(d_pixels, 4, 4, args);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, 16 * sizeof(Pixel<3>), hipMemcpyDeviceToHost);

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(pixels[i], pixels_expected[i]) << "Mismatch at index " << i;
    }
    hipFree(d_pixels);
}

TEST(KernelHelpers, invert_kernel_test) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> pixels_expected[16] = {
        {255, 255, 255}, {0, 0, 0}, {255, 255, 255}, {0, 0, 0},
        {255, 255, 255}, {0, 0, 0}, {255, 255, 255}, {0, 0, 0},
        {255, 255, 255}, {0, 0, 0}, {255, 255, 255}, {0, 0, 0},
        {255, 255, 255}, {0, 0, 0}, {255, 255, 255}, {0, 0, 0}
    };

    Pixel<3> *d_pixels;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);

    struct filter_args args;

    invert_kernel<3><<<1, 1024>>>(d_pixels, 4, 4, args);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, 16 * sizeof(Pixel<3>), hipMemcpyDeviceToHost);

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(pixels[i], pixels_expected[i]) << "Mismatch at index " << i;
    }

    hipFree(d_pixels);
}

TEST(KernelHelpers, clamp_pixels_1) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> expected[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> *h_pixels = new Pixel<3>[16];

    for(int i = 0; i < 16; i++) {
        h_pixels[i] = pixels[i];
        clamp_pixels<3>(&h_pixels[i], i);
    }

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(h_pixels[i], expected[i]) << "Mismatch at index " << i;
    }
}

TEST(KernelHelpers, block_reduce_simple) {

}

TEST(KernelHelpers, block_reduce_randomized) {

}

TEST(KernelHelpers, block_reduce_max) {

}

TEST(KernelHelpers, block_reduce_min) {

}

TEST(KernelHelpers, warp_reduce_simple) {

}

TEST(KernelHelpers, warp_reduce_randomized) {

}

TEST(OtherKernels, image_reduction_simple) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> expected_max = {255, 255, 255};
    Pixel<3> expected_min = {0, 0, 0};

    Pixel<3> real_max, real_min;
    real_max = {SHORT_MIN, SHORT_MIN, SHORT_MIN};
    real_min = {SHORT_MAX, SHORT_MAX, SHORT_MAX}; 

    Pixel<3> *d_pixels;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);

    image_reduction<3>(d_pixels, &real_max, 16, MAX_REDUCE, 1024);
    image_reduction<3>(d_pixels, &real_min, 16, MIN_REDUCE, 1024);

    ASSERT_EQ(real_max, expected_max);
    ASSERT_EQ(real_min, expected_min);

    hipFree(d_pixels);
}

TEST(OtherKernels, image_reduction_randomized) {
    Pixel<3> pixels[16];
    init_image<3>(pixels, 16);

    Pixel<3> expected_max = cpu_image_reduction<3>(pixels, 16, MAX_REDUCE);
    Pixel<3> expected_min = cpu_image_reduction<3>(pixels, 16, MIN_REDUCE);

    Pixel<3> real_max, real_min;
    real_max = {SHORT_MIN, SHORT_MIN, SHORT_MIN};
    real_min = {SHORT_MAX, SHORT_MAX, SHORT_MAX}; 

    Pixel<3> *d_pixels;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);

    image_reduction<3>(d_pixels, &real_max, 16, MAX_REDUCE, 1024);
    image_reduction<3>(d_pixels, &real_min, 16, MIN_REDUCE, 1024);

    ASSERT_EQ(real_max, expected_max);
    ASSERT_EQ(real_min, expected_min);

    hipFree(d_pixels);
}

TEST(ApplyFilter, apply_filter_identity_simple) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> expected[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };
    
    Pixel<3> *output = new Pixel<3>[16];

    filter_args args;
    memset(&args, 0, sizeof(filter_args));
    args.dimension = 3;

    run_kernel<3>("Identity", pixels, output, 4, 4, args);

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(output[i], expected[i]) << "Mismatch at index " << i;
    }
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}