#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "kernel.cuh"
#include "hipcub/hipcub.hpp"
#include "reduce.cuh"
#include "filters.h"
#include "filter_impl.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

const filter** filters = init_filters();

void print_to(const Pixel<3>& pixel, ::std::ostream* os) {
    // recall pixels have short4
    *os << "(" << pixel.data.x << ", " << pixel.data.y << ", " << pixel.data.z << ", " << pixel.data.w << ")";
}

void print_to(const Pixel<4>& pixel, ::std::ostream* os) {
    // recall pixels have short4
    *os << "(" << pixel.data.x << ", " << pixel.data.y << ", " << pixel.data.z << ", " << pixel.data.w << ")";
}

TEST(KernelHelpers, find_index) {
    int index = find_index(0, 0, 3, 3);
    ASSERT_EQ(index, -1);
    index = find_index(3, 3, 0, 0);
    ASSERT_EQ(index, 0);
    index = find_index(3, 3, 1, 1);
    ASSERT_EQ(index, 4);
    index = find_index(3, 3, 2, 2);
    ASSERT_EQ(index, 8);
    index = find_index(3, 3, 2, 1);
    ASSERT_EQ(index, 7);
}

TEST(KernelHelpers, clamp_pixels) {
    Pixel<3> pixel = {-20, -230, 300};
    clamp_pixels<3>(&pixel, 0);
    ASSERT_EQ(pixel.data.x, 0);
    ASSERT_EQ(pixel.data.y, 0);
    ASSERT_EQ(pixel.data.z, 255);

    // create a list of pixels
    Pixel<3> pixels[3] = {{-20, -230, 300}, {0, 0, 0}, {255, 255, 255}};
    clamp_pixels<3>(pixels, 0);
    clamp_pixels<3>(pixels, 1);
    clamp_pixels<3>(pixels, 2);
    ASSERT_EQ(pixels[0].data.x, 0);
    ASSERT_EQ(pixels[0].data.y, 0);
    ASSERT_EQ(pixels[0].data.z, 255);
    ASSERT_EQ(pixels[1].data.x, 0);
    ASSERT_EQ(pixels[1].data.y, 0);
    ASSERT_EQ(pixels[1].data.z, 0);
    ASSERT_EQ(pixels[2].data.x, 255);
    ASSERT_EQ(pixels[2].data.y, 255);
    ASSERT_EQ(pixels[2].data.z, 255);
}

TEST(KernelHelpers, shift_colours) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    // blue values get multiplied by two
    Pixel<3> expected[16] = {
        {0, 0, 0}, {255, 255, 510}, {0, 0, 0}, {255, 255, 510},
        {0, 0, 0}, {255, 255, 510}, {0, 0, 0}, {255, 255, 510},
        {0, 0, 0}, {255, 255, 510}, {0, 0, 0}, {255, 255, 510},
        {0, 0, 0}, {255, 255, 510}, {0, 0, 0}, {255, 255, 510}
    };

    Pixel<3> *d_pixels;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);

    struct filter_args args;
    args.alpha_shift = 0;
    args.red_shift = 0;
    args.green_shift = 0;
    args.blue_shift = 100;

    shift_kernel<3><<<1, 1024>>>(d_pixels, 4, 4, args);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, 16 * sizeof(Pixel<3>), hipMemcpyDeviceToHost);

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(pixels[i], expected[i]) << "Mismatch at index " << i;
    }
    hipFree(d_pixels);
}

TEST(KernelHelpers, brightness_kernel_test) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> pixels_expected[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> *d_pixels;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);

    struct filter_args args;
    args.brightness = 0;

    brightness_kernel<3><<<1, 1024>>>(d_pixels, 4, 4, args);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, 16 * sizeof(Pixel<3>), hipMemcpyDeviceToHost);

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(pixels[i], pixels_expected[i]) << "Mismatch at index " << i;
    }
    hipFree(d_pixels);
}

TEST(KernelHelpers, invert_kernel_test) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> pixels_expected[16] = {
        {255, 255, 255}, {0, 0, 0}, {255, 255, 255}, {0, 0, 0},
        {255, 255, 255}, {0, 0, 0}, {255, 255, 255}, {0, 0, 0},
        {255, 255, 255}, {0, 0, 0}, {255, 255, 255}, {0, 0, 0},
        {255, 255, 255}, {0, 0, 0}, {255, 255, 255}, {0, 0, 0}
    };

    Pixel<3> *d_pixels;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);

    struct filter_args args;

    invert_kernel<3><<<1, 1024>>>(d_pixels, 4, 4, args);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, 16 * sizeof(Pixel<3>), hipMemcpyDeviceToHost);

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(pixels[i], pixels_expected[i]) << "Mismatch at index " << i;
    }

    hipFree(d_pixels);
}

TEST(NormalizationCorrectness, normalize_out_of_bounds) {
    Pixel<4> pixels[4] = {
        {0, 0, 0, 0}, {123, 34, 53, 255},
        {324, 33, -72, 0}, {0, 0, 0, 0}
    };

    Pixel<4> expected[4] = {
        {0, 0, 0, 0}, {96, 255, 255, 255},
        {255, 247, 0, 0}, {0, 0, 0, 0}
    };

    Pixel<4> *h_output = new Pixel<4>[4];

    Pixel<4> *d_pixels = nullptr;
    hipMalloc(&d_pixels, 4 * sizeof(Pixel<4>));
    hipMemcpy(d_pixels, pixels, 4 * sizeof(Pixel<4>), hipMemcpyHostToDevice);

    Pixel<4> h_smallest = {0, 0, -72, 0};
    Pixel<4> h_largest = {324, 34, 53, 255};

    Pixel<4> *d_smallest = nullptr;
    hipMalloc(&d_smallest, sizeof(Pixel<4>));
    Pixel<4> *d_largest = nullptr;
    hipMalloc(&d_largest, sizeof(Pixel<4>));

    hipMemcpy(d_smallest, &h_smallest, sizeof(Pixel<4>), hipMemcpyHostToDevice);
    hipMemcpy(d_largest, &h_largest, sizeof(Pixel<4>), hipMemcpyHostToDevice);

    normalize<4><<<1, 1024>>>(d_pixels, 2, 2, d_smallest, d_largest, true);
    hipDeviceSynchronize();
    hipMemcpy(h_output, d_pixels, 4 * sizeof(Pixel<4>), hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();

    for(int i = 0; i < 4; i++) {
        ASSERT_EQ(expected[i], h_output[i]) << "Mismatch at index " << i;
    }

    hipFree(d_pixels);
    hipFree(d_smallest);
    hipFree(d_largest);
}  

int main(int argc, char **argv) {
    setenv("current_dir", getenv("PWD"), 1);
    const char* current_dir = getenv("current_dir");
    if(current_dir != NULL) {
        // set it one layer outside i.e ../
        char *parent_dir = new char[strlen(current_dir) + 3];
        strcpy(parent_dir, current_dir);
        strcat(parent_dir, "/..");
        // now set this to current_dir
        setenv("current_dir", parent_dir, 1);
    }
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}