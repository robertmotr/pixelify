#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "filters.h"
#include "filter_impl.h"
#include "gtest/gtest.h"
#include <string>

#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wunused-but-set-variable"

#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"

#pragma GCC diagnostic pop

const filter** filters = init_filters();

void print_to(const Pixel<3>& pixel, ::std::ostream* os) {
    // recall pixels have short4
    *os << "(" << pixel.data.x << ", " << pixel.data.y << ", " << pixel.data.z << ", " << pixel.data.w << ")";
}

void print_to(const Pixel<4>& pixel, ::std::ostream* os) {
    // recall pixels have short4
    *os << "(" << pixel.data.x << ", " << pixel.data.y << ", " << pixel.data.z << ", " << pixel.data.w << ")";
}

// Helper function to initialize an array of pixels
template<unsigned int channels>
void init_image(Pixel<channels> *d_image, int pixels) {
    for (int i = 0; i < pixels; ++i) {
        for (int channel = 0; channel < channels; ++channel) {
            d_image[i].set(channel, rand() % 256);
        }
    }
}

// CPU reduction function for verification
template <unsigned int channels>
Pixel<channels> cpu_image_reduction(const Pixel<channels> *image, int pixels, bool reduce_type) {
    Pixel<channels> result;
    for (int channel = 0; channel < channels; ++channel) {
        if (reduce_type == MAX_REDUCE) {
            result.set(channel, SHORT_MAX);
        } else {
            result.set(channel, SHORT_MIN);
        }
    }

    for (int i = 0; i < pixels; ++i) {
        for (int channel = 0; channel < channels; ++channel) {
            if (reduce_type == MAX_REDUCE) {
                result.set(channel, max(result.at(channel), image[i].at(channel)));
            } else {
                result.set(channel, min(result.at(channel), image[i].at(channel)));
            }
        }
    }
    return result;
}

TEST(KernelHelpers, find_index) {
    int index = find_index(0, 0, 3, 3);
    ASSERT_EQ(index, -1);
    index = find_index(3, 3, 0, 0);
    ASSERT_EQ(index, 0);
    index = find_index(3, 3, 1, 1);
    ASSERT_EQ(index, 4);
    index = find_index(3, 3, 2, 2);
    ASSERT_EQ(index, 8);
    index = find_index(3, 3, 2, 1);
    ASSERT_EQ(index, 7);
}

TEST(KernelHelpers, clamp_pixels) {
    Pixel<3> pixel = {-20, -230, 300};
    clamp_pixels<3>(&pixel, 0);
    ASSERT_EQ(pixel.data.x, 0);
    ASSERT_EQ(pixel.data.y, 0);
    ASSERT_EQ(pixel.data.z, 255);

    // create a list of pixels
    Pixel<3> pixels[3] = {{-20, -230, 300}, {0, 0, 0}, {255, 255, 255}};
    clamp_pixels<3>(pixels, 0);
    clamp_pixels<3>(pixels, 1);
    clamp_pixels<3>(pixels, 2);
    ASSERT_EQ(pixels[0].data.x, 0);
    ASSERT_EQ(pixels[0].data.y, 0);
    ASSERT_EQ(pixels[0].data.z, 255);
    ASSERT_EQ(pixels[1].data.x, 0);
    ASSERT_EQ(pixels[1].data.y, 0);
    ASSERT_EQ(pixels[1].data.z, 0);
    ASSERT_EQ(pixels[2].data.x, 255);
    ASSERT_EQ(pixels[2].data.y, 255);
    ASSERT_EQ(pixels[2].data.z, 255);
}

TEST(KernelHelpers, shift_colours) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    // blue values get multiplied by two
    Pixel<3> expected[16] = {
        {0, 0, 0}, {255, 255, 510}, {0, 0, 0}, {255, 255, 510},
        {0, 0, 0}, {255, 255, 510}, {0, 0, 0}, {255, 255, 510},
        {0, 0, 0}, {255, 255, 510}, {0, 0, 0}, {255, 255, 510},
        {0, 0, 0}, {255, 255, 510}, {0, 0, 0}, {255, 255, 510}
    };

    Pixel<3> *d_pixels;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);

    struct filter_args args;
    args.alpha_shift = 0;
    args.red_shift = 0;
    args.green_shift = 0;
    args.blue_shift = 100;

    shift_kernel<3><<<1, 1024>>>(d_pixels, 4, 4, args);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, 16 * sizeof(Pixel<3>), hipMemcpyDeviceToHost);

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(pixels[i], expected[i]) << "Mismatch at index " << i;
    }
    hipFree(d_pixels);
}

TEST(KernelHelpers, brightness_kernel_test) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> pixels_expected[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> *d_pixels;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);

    struct filter_args args;
    args.brightness = 0;

    brightness_kernel<3><<<1, 1024>>>(d_pixels, 4, 4, args);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, 16 * sizeof(Pixel<3>), hipMemcpyDeviceToHost);

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(pixels[i], pixels_expected[i]) << "Mismatch at index " << i;
    }
    hipFree(d_pixels);
}

TEST(KernelHelpers, invert_kernel_test) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> pixels_expected[16] = {
        {255, 255, 255}, {0, 0, 0}, {255, 255, 255}, {0, 0, 0},
        {255, 255, 255}, {0, 0, 0}, {255, 255, 255}, {0, 0, 0},
        {255, 255, 255}, {0, 0, 0}, {255, 255, 255}, {0, 0, 0},
        {255, 255, 255}, {0, 0, 0}, {255, 255, 255}, {0, 0, 0}
    };

    Pixel<3> *d_pixels;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);

    struct filter_args args;

    invert_kernel<3><<<1, 1024>>>(d_pixels, 4, 4, args);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_pixels, 16 * sizeof(Pixel<3>), hipMemcpyDeviceToHost);

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(pixels[i], pixels_expected[i]) << "Mismatch at index " << i;
    }

    hipFree(d_pixels);
}

TEST(KernelHelpers, clamp_pixels_1) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> expected[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> *h_pixels = new Pixel<3>[16];

    for(int i = 0; i < 16; i++) {
        h_pixels[i] = pixels[i];
        clamp_pixels<3>(&h_pixels[i], i);
    }

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(h_pixels[i], expected[i]) << "Mismatch at index " << i;
    }
}

TEST(OtherKernels, image_reduction_simple) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    std::cout << "before cudamalloc" << std::endl;

    Pixel<3> *d_pixels = nullptr;
    std::cout << "did we get here?" << std::endl;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    CUDA_CHECK_ERROR("malloc");
    std::cout << "maybe the malloc is crashing" << std::endl;
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR("memcpy");

    std::cout << "after cudamalloc" << std::endl;

    Pixel<3> *d_expected_max, *d_expected_min;
    Pixel<3> *h_expected_max, *h_expected_min;
    hipMalloc(&d_expected_max, sizeof(Pixel<3>));
    hipMalloc(&d_expected_min, sizeof(Pixel<3>));

    std::cout << "this means we havent gotten to cpu image reduction yet" << std::endl;

    // assert cpu image reduction is correct
    Pixel<3> expected_max = cpu_image_reduction<3>(pixels, 16, MAX_REDUCE);
    Pixel<3> expected_min = cpu_image_reduction<3>(pixels, 16, MIN_REDUCE);

    std::cout << "see if we made it here PART 1" << std::endl;

    image_reduction<3>(d_pixels, d_expected_max, 16, MAX_REDUCE);
    image_reduction<3>(d_pixels, d_expected_min, 16, MIN_REDUCE);

    h_expected_max = new Pixel<3>;
    h_expected_min = new Pixel<3>;

    hipMemcpy(h_expected_max, d_expected_max, sizeof(Pixel<3>), hipMemcpyDeviceToHost);
    hipMemcpy(h_expected_min, d_expected_min, sizeof(Pixel<3>), hipMemcpyDeviceToHost);

    std::cout << "see if we made it here PART 2" << std::endl;

    for(int i = 0; i < 3; i++) {
        ASSERT_EQ(h_expected_max->at(i), 255);
        ASSERT_EQ(h_expected_min->at(i), 0);
    }

    ASSERT_EQ(*h_expected_max, expected_max);
    ASSERT_EQ(*h_expected_min, expected_min);

    std::cout << "see if we made it here PART 3" << std::endl;

    hipFree(d_pixels);
    hipFree(d_expected_max);
    hipFree(d_expected_min);
    delete h_expected_max;
    delete h_expected_min;
}

TEST(OtherKernels, image_reduction_randomized) {
    Pixel<3> pixels[16];
    init_image<3>(pixels, 16);

    Pixel<3> expected_max = cpu_image_reduction<3>(pixels, 16, MAX_REDUCE);
    Pixel<3> expected_min = cpu_image_reduction<3>(pixels, 16, MIN_REDUCE);

    std::cout << "crash here? expected max: " << std::endl;

    Pixel<3> real_max, real_min;
    real_max = {SHORT_MIN, SHORT_MIN, SHORT_MIN};
    real_min = {SHORT_MAX, SHORT_MAX, SHORT_MAX}; 

    Pixel<3> *d_pixels;
    hipMalloc(&d_pixels, 16 * sizeof(Pixel<3>));
    hipMemcpy(d_pixels, pixels, 16 * sizeof(Pixel<3>), hipMemcpyHostToDevice);

    image_reduction<3>(d_pixels, &real_max, 16, MAX_REDUCE);
    image_reduction<3>(d_pixels, &real_min, 16, MIN_REDUCE);

    ASSERT_EQ(real_max, expected_max);
    ASSERT_EQ(real_min, expected_min);

    hipFree(d_pixels);
}

TEST(ApplyFilter, apply_filter_identity_simple) {
    Pixel<3> pixels[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };

    Pixel<3> expected[16] = {
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255},
        {0, 0, 0}, {255, 255, 255}, {0, 0, 0}, {255, 255, 255}
    };
    
    Pixel<3> *output = new Pixel<3>[16];

    filter_args args;
    memset(&args, 0, sizeof(filter_args));
    args.dimension = 3;

    run_kernel<3>("Identity", pixels, output, 4, 4, args);

    for(int i = 0; i < 16; i++) {
        ASSERT_EQ(output[i], expected[i]) << "Mismatch at index " << i;
    }
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}