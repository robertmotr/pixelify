#include "hip/hip_runtime.h"
#include "reduce.h"

__global__ void reduce_max(int *in, int* out, unsigned int N)
{
    int max_val = INT_MIN; 

    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        max_val = max(max_val, in[i]);
    }

    max_val = blockReduceMax(max_val);

    if (threadIdx.x == 0)
        atomicMax(out, max_val);
}

__global__ void reduce_min(int *in, int* out, unsigned int N)
{
    int min_val = INT_MAX;

    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        min_val = min(min_val, in[i]);
    }

    min_val = blockReduceMin(min_val);

    if (threadIdx.x == 0)
        atomicMin(out, min_val);
}