#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "reduce.h"
#include "pixel.h"
#include "filter_impl.h"
#include "filters.h"
#include "sharedmem.cuh"
#include <hip/hip_runtime.h>

template<unsigned int channels>
void run_kernel(const char *filter_name, const Pixel<channels> *input,
                 Pixel<channels> *output, int width, int height, struct kernel_args extra) {
  
  const size_t src_pitch =                               width * sizeof(Pixel<channels>);
  const filter *h_filter =                               nullptr;
  filter*                                                device_filter;
  int*                                                   device_filter_data;
  char*                                                  device_filter_name;
  int                                                    pixels = width * height;
  Pixel<channels>                                        *device_output;
  Pixel<channels>                                        *d_largest, *d_smallest;
  Pixel<channels>                                        *h_pinned_input, *h_pinned_output;
  Pixel<channels>                                        *h_smallest, *h_largest;          
  int blockSize;
  int gridSize;
  hipArray_t cu_array;
  hipTextureObject_t tex_obj =                          0;
  h_smallest =                                           new Pixel<channels>(SHORT_MAX);
  h_largest =                                            new Pixel<channels>(SHORT_MIN);

  if(strcmp(filter_name, "NULL") != 0) {         
    h_filter = create_filter(filter_name, extra.dimension, extra.filter_strength);
    if(h_filter == nullptr) {
      printf("Error: filter is null\n");
      exit(1);
    }
  } 

  hipDeviceGetAttribute(&blockSize, hipDeviceAttributeMaxThreadsPerBlock, 0);
  assert(blockSize != 0);
  gridSize = (8 * height + blockSize - 1) / blockSize; 

  // create copy of input, output on pinned memory on host
  hipHostAlloc(&h_pinned_input, pixels * sizeof(Pixel<channels>), hipHostMallocDefault);
  hipHostAlloc(&h_pinned_output, pixels * sizeof(Pixel<channels>), hipHostMallocDefault); // possible bug
  hipMemcpy(h_pinned_input, input, pixels * sizeof(Pixel<channels>), hipMemcpyHostToHost);
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("copying input to pinned input");

  // MALLOCS ON DEVICE
  hipMalloc(&device_output, pixels * sizeof(Pixel<channels>));
  hipMalloc(&d_largest, sizeof(Pixel<channels>));
  hipMalloc(&d_smallest, sizeof(Pixel<channels>));
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("cuda mallocs for input, output, largest, smallest");

  hipChannelFormatDesc channel_desc = hipCreateChannelDesc(8 * sizeof(short), 
                                                             8 * sizeof(short), 
                                                             8 * sizeof(short),
                                                             8 * sizeof(short), 
                                                             hipChannelFormatKindSigned); 

  hipMallocArray(&cu_array, &channel_desc, width, height);
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("cuda malloc array");

  hipMemcpy2DToArray(cu_array, 0, 0, h_pinned_input, src_pitch, width * sizeof(Pixel<channels>), (size_t) height, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("2d array copy to device_input");

  struct hipResourceDesc res_desc;
  memset(&res_desc, 0, sizeof(res_desc));
  res_desc.resType = hipResourceTypeArray;
  res_desc.res.array.array = cu_array;
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("resource desc");

  struct hipTextureDesc tex_desc;
  memset(&tex_desc, 0, sizeof(tex_desc));
  tex_desc.addressMode[0] = hipAddressModeBorder;
  tex_desc.addressMode[1] = hipAddressModeBorder;
  tex_desc.filterMode = hipFilterModePoint;
  tex_desc.readMode = hipReadModeElementType;
  tex_desc.normalizedCoords = 0;
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("texture desc");

  // texture object
  hipCreateTextureObject(&tex_obj, &res_desc, &tex_desc, NULL);
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("creating texture object");

  // HANDLE MALLOC AND MEMCPY FOR FILTER ONLY
  if(h_filter != nullptr && strcmp(filter_name, "NULL") != 0) {
    hipMalloc(&device_filter, sizeof(filter));
    hipMemcpy(&(device_filter->filter_dimension), &(h_filter->filter_dimension), sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(&(device_filter->name_size), &(h_filter->name_size), sizeof(size_t), hipMemcpyHostToDevice);

    hipMalloc(&device_filter_data, h_filter->filter_dimension * h_filter->filter_dimension * sizeof(unsigned int));
    hipMemcpy(device_filter_data, h_filter->filter_data, h_filter->filter_dimension * h_filter->filter_dimension * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&(device_filter->filter_data), &device_filter_data, sizeof(float*), hipMemcpyHostToDevice);

    hipMalloc(&device_filter_name, h_filter->name_size * sizeof(char));
    hipMemcpy(device_filter_name, h_filter->filter_name, h_filter->name_size * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(&(device_filter->filter_name), &device_filter_name, sizeof(char*), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("cuda mallocs and memcpies for filter");
  }

  // MEMCPYS FROM HOST TO DEVICE
  hipMemcpy(d_smallest, h_smallest, sizeof(Pixel<channels>), hipMemcpyHostToDevice);
  hipMemcpy(d_largest, h_largest, sizeof(Pixel<channels>), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("cuda memcpys and mallocs");

  // apply filter first if filter is not NULL
  // then apply everything else in the kernel_args struct
  // but first apply it filter_passes times
  for(int pass = 0; pass < extra.passes; pass++) {
    filter_kernel<channels><<<gridSize, blockSize, sizeof(float) * h_filter->filter_dimension * h_filter->filter_dimension>>>(tex_obj, device_output,
                                                                                      width, height, device_filter, extra);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("filter kernel");
    hipMemcpy2DToArray(cu_array, 0, 0, device_output, src_pitch, src_pitch, (size_t) height, hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("2d array copy back to device_output");
  }
  // then apply everything else in the kernel_args struct
  if(extra.alpha_shift != 0 || extra.red_shift != 0 || extra.green_shift != 0 || extra.blue_shift != 0) {
    other_kernel<channels><<<gridSize, blockSize, blockSize * sizeof(Pixel<channels>)>>>(device_output, device_output, width, height, OP_SHIFT_COLOURS, extra);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("shift colours");
    hipMemcpy2DToArray(cu_array, 0, 0, device_output, src_pitch, src_pitch, (size_t) height, hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("2d array copy back to device_output");
  }
  if(extra.tint[0] != 0 || extra.tint[1] != 0 || extra.tint[2] != 0 || extra.tint[3] != 0) {
    other_kernel<channels><<<gridSize, blockSize, blockSize * sizeof(Pixel<channels>)>>>(device_output, device_output, width, height, OP_TINT, extra);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("tint");
    hipMemcpy2DToArray(cu_array, 0, 0, device_output, src_pitch, src_pitch, (size_t) height, hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("2d array copy back to device_output");
  }
  if(extra.brightness != 0) {
    other_kernel<channels><<<gridSize, blockSize, blockSize * sizeof(Pixel<channels>)>>>(device_output, device_output, width, height, OP_BRIGHTNESS, extra);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("brightness");
    hipMemcpy2DToArray(cu_array, 0, 0, device_output, src_pitch, src_pitch, (size_t) height, hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("2d array copy back to device_output");
  }

  // parallel reduction to find largest and smallest pixel values
  // for each channel respectively
  image_reduction<channels>(device_output, d_largest, pixels, MAX_REDUCE);
  image_reduction<channels>(device_output, d_smallest, pixels, MIN_REDUCE);

  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("reduction");

  // if d_largest or d_smallest are out of bounds
  // i.e outside of [0, 255] for any channel
  // then we need to normalize the image to bring it into valid bounds
  hipMemcpy(h_smallest, d_smallest, sizeof(Pixel<channels>), hipMemcpyDeviceToHost);
  hipMemcpy(h_largest, d_largest, sizeof(Pixel<channels>), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("copying back d_smallest and d_largest");
  
  for(int ch = 0; ch < channels; ch++) {
    if(h_smallest->data[ch] < 0 || h_smallest->data[ch] > 255 ||
       h_largest->data[ch] < 0 || h_largest->data[ch] > 255) {
        normalize<channels><<<gridSize, blockSize>>>(device_output, width, height, d_smallest, d_largest, extra.normalize);
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("normalize");
        break;
    }
  }

  hipMemcpy(h_pinned_output, device_output, pixels * sizeof(Pixel<channels>), hipMemcpyDeviceToHost);
  hipMemcpy(output, h_pinned_output, pixels * sizeof(Pixel<channels>), hipMemcpyHostToHost);
  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("copying back d_output to pinned output");
  
  // cleanup
  hipDestroyTextureObject(tex_obj);
  hipFreeArray(cu_array);

  hipHostFree(h_pinned_input); hipHostFree(h_pinned_output);
  delete h_smallest;
  delete h_largest;
  hipFree(device_filter);
  hipFree(d_smallest); hipFree(d_largest);
  hipFree(device_output);
  
  if(!(h_filter->properties->basic_filter)) {
    delete h_filter; // only delete if its NOT a basic filter
    // basic filters get reused
  }

  hipDeviceSynchronize();
  CUDA_CHECK_ERROR("freeing memory");
}

template<unsigned int channels>
__global__ void filter_kernel(const hipTextureObject_t tex_obj, Pixel<channels> *out, int width, int height,
                              const filter *filter, const struct kernel_args args) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int total_threads = blockDim.x * gridDim.x;

  extern __shared__ float smem[];
  for(int i = 0; i < filter->filter_dimension * filter->filter_dimension; i++) {
    smem[i] = filter->filter_data[i];
  }
  __syncthreads();

  #pragma unroll
  for(int pixel_idx = tid; pixel_idx < width * height; pixel_idx += total_threads) {

    int row = pixel_idx / width;
    int col = pixel_idx % width;

    #pragma unroll
    for(int ch = 0; ch < channels; ch++) {
      out[pixel_idx].data[ch] = apply_filter<channels>(tex_obj, filter, ch, width, height, row, col);
    }  
  } 
}

template<unsigned int channels>
__global__  void other_kernel(const Pixel<channels> *in, Pixel<channels> *out, int width, int height,
                              unsigned char operation, struct kernel_args extra) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int total_threads = blockDim.x * gridDim.x;

  // load data and apply operation at same time
  #pragma unroll
  for(int pixel_idx = tid; pixel_idx < width * height; pixel_idx += total_threads) {
    
    #pragma unroll
    for(int channel = 0; channel < channels; channel++) {

      // if(operation == OP_SHIFT_COLOURS) {
      //   out[pixel_idx].data[channel] = shift_colours(smem_other[threadIdx.x].data[channel], extra, channel);
      // } else if(operation == OP_BRIGHTNESS) {
      //   out[pixel_idx].data[channel] = smem_other[threadIdx.x].data[channel] * (100 + extra.brightness) / 100;
      // }
      // else if(operation == OP_TINT) {
      //   out[pixel_idx].data[channel] = (1 - (float)(extra.blend_factor / 100)) * extra.tint[channel] + 
      //                                     (float)(extra.blend_factor / 100) * smem_other[pixel_idx].data[channel];
      // }
    }
  }
}

template<unsigned int channels>
__global__ void normalize(Pixel<channels> *target, int width, int height,
                           const Pixel<channels> *smallest, const Pixel<channels> *largest, bool normalize_or_clamp) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int total_threads = blockDim.x * gridDim.x;
  
  #pragma unroll
  for(int pixel_idx = tid; pixel_idx < width * height; pixel_idx += total_threads) {
    if(normalize_or_clamp) {
      normalize_pixel<channels>(target, pixel_idx, smallest, largest);
    } else {
      clamp_pixels<channels>(target, pixel_idx);
    }
  }
}

// EXPLICIT INSTANTIATIONS:
template void run_kernel(const char *filter_name, const Pixel<3u> *input,
                 Pixel<3u> *output, int width, int height, struct kernel_args extra);

template void run_kernel(const char *filter_name, const Pixel<4u> *input, 
                 Pixel<4u> *output, int width, int height, struct kernel_args extra);

