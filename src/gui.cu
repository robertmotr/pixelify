#include "gui.h"
#include "stb_image.h"
#include "stb_image_write.h"

#include "filters.h"
#include "filter_impl.h"
#include "kernel_formulas.h"

#include <GL/gl.h>  // or #include <GL/glew.h>
#include "tex_inspect_opengl.h"
#include "imgui_tex_inspect.h"
#include "imgui_tex_inspect_internal.h"

#include <hip/hip_runtime.h>

inline void display_image(const GLuint& texture, const int& width, const int& height,
                          const unsigned char *image_data) {

    ImGui::Text("size = %d x %d", width, height);
    ImGuiIO& io = ImGui::GetIO();
    ImVec2 pos = ImGui::GetCursorScreenPos();
    if(ImGuiTexInspect::BeginInspectorPanel("##IMAGE", (void*)(intptr_t)texture, ImVec2(width, height), 
                                            ImGuiTexInspect::InspectorFlags_FillHorizontal | ImGuiTexInspect::InspectorFlags_FillVertical)) {
        ImGuiTexInspect::DrawAnnotations(ImGuiTexInspect::ValueText(ImGuiTexInspect::ValueText::BytesDec));
    }   
    ImGuiTexInspect::EndInspectorPanel();
    struct ImGuiTexInspect::Context *ctx = ImGuiTexInspect::GetContext();
    struct ImGuiTexInspect::Inspector *inspector = ctx->CurrentInspector;

    // Check if the mouse is within the bounds of the image
    if (ImGui::IsMouseHoveringRect(pos, inspector->PanelSize)) {

        ImGui::BeginTooltip();
        ImGui::Text("Hold left-click to inspect the image.\nYou can also zoom in/out and pan the image.");
        ImGui::EndTooltip();

        // check if user is holding left ctrl key
        if(io.KeyCtrl) {
            ImVec2 mouse_pos = ImGui::GetMousePos();

            ImVec2 panpos = inspector->PanPos;
            ImVec2 scale = inspector->Scale;
            ImVec2 topleft = inspector->PanelTopLeftPixel;
            ImVec2 panelsize = inspector->PanelSize;
            ImVec2 viewsize = inspector->ViewSize;
            ImVec2 viewsizeuv = inspector->ViewSizeUV;

            // Calculate the UV coordinates corresponding to the mouse position
            ImVec2 uv = ImVec2((mouse_pos.x - pos.x) / width, (mouse_pos.y - pos.y) / height);

            // Calculate the texel coordinates based on the inspector state
            ImVec2 texel_coordinates = ImVec2((uv.x - panpos.x) / scale.x, (uv.y - panpos.y) / scale.y);

            // Now you can use texel_coordinates to inspect the image or perform any other actions
            // For example, you might want to pass texel_coordinates to your ImageInspect::inspect function
            ImageInspect::inspect(width, height, image_data, texel_coordinates, viewsize);
        }
    }
}


inline void display_tab_bar(bool& original_loaded, bool& preview_loaded, const int& width, const int& height, 
                            const GLuint& texture_orig, const GLuint& texture_preview, const unsigned char *image_orig, 
                            const unsigned char *image_preview) { 

    if (ImGui::BeginTabBar("tab_bar", ImGuiTabBarFlags_None)) {
        if (ImGui::BeginTabItem("Original image")) {
            if(original_loaded) {
                display_image(texture_orig, width, height, image_orig);
            }
            else {
                ImGui::Text("No image loaded. Please select an input file and a filter on the right side panel, and click Apply Changes.");
            }
            ImGui::EndTabItem();
        }
        ImGui::SetNextItemWidth(200.0f);
        if (ImGui::BeginTabItem("Preview transformations")) {
            if(preview_loaded) {
                display_image(texture_preview, width, height, image_preview);
            }
            else {
                ImGui::Text("No image loaded. Please select an input file and a filter on the right side panel, and click Apply Changes.");
            }
            ImGui::EndTabItem();
        }
        ImGui::SetNextItemWidth(200.0f);
        if (ImGui::BeginTabItem("Settings")) {
            ImGui::Text("TODO: add settings");   // TODO
            ImGui::Text("But, to be honest, theres not much to add here.");
            ImGui::EndTabItem();
        }
        ImGui::SetNextItemWidth(200.0f);
        if(ImGui::BeginTabItem("Analytics")) {
            ImGui::Text("TODO: add analytics");  // TODO
            // add graphs, charts, etc
            ImGui::EndTabItem();
        }
        ImGui::EndTabBar();
    }
}

std::string generate_exif_string(const Exiv2::ExifData& exifData) {

    if(exifData.empty()) {
        return std::string("No EXIF data found in file\n");
    }

    std::ostringstream result;

    Exiv2::ExifData::const_iterator end = exifData.end();
    for (Exiv2::ExifData::const_iterator i = exifData.begin(); i != end; ++i) {
        const char* tn = i->typeName();
        result << std::setw(44) << std::setfill(' ') << std::left
               << i->key() << " "
               << "0x" << std::setw(4) << std::setfill('0') << std::right
               << std::hex << i->tag() << " "
               << std::setw(9) << std::setfill(' ') << std::left
               << (tn ? tn : "Unknown") << " "
               << std::dec << std::setw(3)
               << std::setfill(' ') << std::right
               << i->count() << "  "
               << std::dec << i->value()
               << "\n";
    }
    return result.str();
}

std::string generate_iptc_string(const Exiv2::IptcData& iptcData) {
    if(iptcData.empty()) {
        return std::string("No IPTC data found in file\n");
    }

    std::ostringstream result;

    auto end = iptcData.end();
    for (auto md = iptcData.begin(); md != end; ++md) {
        result << std::setw(44) << std::setfill(' ') << std::left
               << md->key() << " "
               << "0x" << std::setw(4) << std::setfill('0') << std::right
               << std::hex << md->tag() << " "
               << std::setw(9) << std::setfill(' ') << std::left
               << md->typeName() << " "
               << std::dec << std::setw(3)
               << std::setfill(' ') << std::right
               << md->count() << "  "
               << std::dec << md->value()
               << std::endl;
    }

    return result.str();
}

std::string generate_xmp_string(const Exiv2::XmpData& xmp_data) {
    if(xmp_data.empty()) {
        return std::string("No XMP data found in file\n");
    }

    std::ostringstream result;

    auto end = xmp_data.end();
    for (auto md = xmp_data.begin(); md != end; ++md) {
     result << std::setfill(' ') << std::left
            << std::setw(44)
            << md->key() << " "
            << std::setw(9) << std::setfill(' ') << std::left
            << md->typeName() << " "
            << std::dec << std::setw(3)
            << std::setfill(' ') << std::right
            << md->count() << "  "
            << std::dec << md->value()
            << std::endl;
    }
    return result.str();
}

void show_ui(ImGuiIO& io) {
    // to determine which tab is shown
    static bool show_original =                 false;
    static bool show_preview =                  false;
    static bool show_tint =                     false;

    // filter options
    struct filter_args                          extra_args;
    static bool normalize =                     false;
    static int passes =                         1;
    static int filter_size =                    3;
    static int filter_strength =                0;
    static int red_strength =                   0;
    static int green_strength =                 0;
    static int blue_strength =                  0;
    static int alpha_strength =                 0;
    static int brightness =                     0;
    static ImVec4 tint_colour =                 ImVec4(0.0f, 0.0f, 0.0f, 0.0f);
    static float blend_factor =                 0;
    static bool invert =                        false;
    static bool conversion =                    false;
    static bool threshold =                     false;

    // image details stuff
    static char input[256] =                    "";
    static char output[256] =                   "";
    static Exiv2::Image::UniquePtr              image;
    static std::string                          exif_data_str;
    static std::string                          iptc_data_str;
    static std::string                          xmp_data_str;

    // rendering stuff
    static int width, height, channels;
    static GLuint texture_orig =                0;
    static GLuint texture_preview =             0;

    // backend stuff
    static unsigned char *image_data =          NULL;
    static unsigned char *image_data_out =      NULL;
    static const filter** filters =             init_filters();
    static int current_filter_dropdown_idx =    0;
    static ImGuiComboFlags flags =              0;
    static filter* selected_filter =            const_cast<filter*>(filters[current_filter_dropdown_idx]);   
    static void *pixels_in =                    NULL;
    static void *pixels_out =                   NULL;
    // static vector<analytic> analytics;

    ImGui::Begin("Workshop", nullptr, ImGuiWindowFlags_NoResize
     | ImGuiWindowFlags_NoMove | ImGuiWindowFlags_NoCollapse | ImGuiWindowFlags_HorizontalScrollbar
     | ImGuiWindowFlags_AlwaysHorizontalScrollbar);

    ImVec2 main_panel_size = ImVec2(2 * ImGui::GetContentRegionAvail().x / 3,
                                     ImGui::GetContentRegionAvail().y - 75);
    ImVec2 side_panel_1_size = ImVec2(ImGui::GetContentRegionAvail().x / 3,
                                     (2 * ImGui::GetContentRegionAvail().y - 80) / 3);
    ImVec2 side_panel_2_size = ImVec2(ImGui::GetContentRegionAvail().x / 3,
                                     (ImGui::GetContentRegionAvail().y - 80) / 3 - 22);

    ImGui::SetWindowSize(main_panel_size);
    ImVec2 parent_cursor_start = ImGui::GetCursorPos();
    ImGui::BeginChild("Main panel", main_panel_size, ImGuiChildFlags_None, ImGuiWindowFlags_AlwaysHorizontalScrollbar);
    ImGui::SetNextItemWidth(200.0f);
    display_tab_bar(show_original, show_preview, width, height, texture_orig,
                    texture_preview, image_data, image_data_out);
    ImGui::EndChild();
    ImGui::SetCursorPos(ImVec2(main_panel_size.x + 10, parent_cursor_start.y));
    ImGui::BeginChild("Side panel 1", side_panel_1_size, true);
    ImGui::InputTextWithHint("Input file path", "Absolute path of your input image", input, IM_ARRAYSIZE(input));
    ImGui::Spacing();

    if (ImGui::Button("Select input file")) {
        IGFD::FileDialogConfig config;
        config.sidePaneWidth = 300.0f;
        config.path = ".";
        ImGuiFileDialog::Instance()->OpenDialog("ChooseFileDlgKey", "Choose File", ".png, .jpg", config);
    }

    if (ImGuiFileDialog::Instance()->Display("ChooseFileDlgKey")) {
        if (ImGuiFileDialog::Instance()->IsOk()) {
            std::string file_path_name = ImGuiFileDialog::Instance()->GetFilePathName();
            std::string file_path = ImGuiFileDialog::Instance()->GetCurrentPath();
            sprintf(input, "%s", file_path_name.c_str());
        }
        ImGuiFileDialog::Instance()->Close();
    }
    ImGui::SameLine();
    // process file path image if user clicks button
    if(ImGui::Button("Open input file")) {
        if(load_texture_from_file(input, &texture_orig, &image_data, &width, &height, &channels) == false) {
            ImGui::OpenPopup("Error loading image");
            show_original = false;
        } else {

            image_data_out = stbi_load(input, &width, &height, &channels, 4);
            if(image_data_out == NULL) {
                printf("Error loading copy of image\n");
                return;
            }

            if(channels == 3) {
                Pixel<3> *px_in = new Pixel<3>[width * height];
                Pixel<3> *px_out = new Pixel<3>[width * height];
                pixels_in = (void*)px_in;
                pixels_out = (void*)px_out;
                // convert image data to pixel array
                imgui_get_pixels<3>(image_data, px_in, width * height);
            }  
            else if(channels == 4) {
                Pixel<4> *px_in = new Pixel<4>[width * height];
                Pixel<4> *px_out = new Pixel<4>[width * height];
                pixels_in = (void*)px_in;
                pixels_out = (void*)px_out;
                // convert image data to pixel array
                imgui_get_pixels<4>(image_data, px_in, width * height);
            }
            else {
                printf("Error: unsupported number of channels\n");
                return;
            }

            image = Exiv2::ImageFactory::open(input);
            assert(image.get() != 0);
            image->readMetadata();
            Exiv2::ExifData& exifdata = image->exifData();
            exif_data_str = generate_exif_string(exifdata);

            Exiv2::IptcData& iptcdata = image->iptcData();
            iptc_data_str = generate_iptc_string(iptcdata);

            Exiv2::XmpData& xmpdata = image->xmpData();
            xmp_data_str = generate_xmp_string(xmpdata);

            show_original = true;
        }
    }
    ImGui::SameLine();
    if(ImGui::Button("Clear original image")) {

        if(channels == 3) {
            if(pixels_in != NULL) {
                delete[] (Pixel<3>*)pixels_in;
                pixels_in = NULL;
            }
            if(pixels_out != NULL) {
                delete[] (Pixel<3>*)pixels_out;
                pixels_out = NULL;
            }
        }
        else if(channels == 4) {
            if(pixels_in != NULL) {
                delete[] (Pixel<4>*)pixels_in;
                pixels_in = NULL;
            }
            if(pixels_out != NULL) {
                delete[] (Pixel<4>*)pixels_out;
                pixels_out = NULL;
            }
        }

        stbi_image_free(image_data);
        stbi_image_free(image_data_out);

        show_original = false;
        show_preview = false;

        if(texture_orig != 0) {
            glDeleteTextures(1, &texture_orig);
            texture_orig = 0;
        }
        if(texture_preview != 0) {
            glDeleteTextures(1, &texture_preview);
            texture_preview = 0;
        }
        printf("Cleared original + preview image successfully.\n");
    }

    if(ImGui::BeginPopup("Error loading image")) {
        ImGui::Text("Error loading image, select a valid path");
        if(ImGui::Button("OK")) {
            ImGui::CloseCurrentPopup();
        }
        ImGui::EndPopup();
    }

    ImGui::Spacing();
    ImGui::Spacing();
    ImGui::InputTextWithHint("##output", "Absolute path for your output image", output, IM_ARRAYSIZE(output));
    ImGui::SameLine();
    ImGui::Spacing();
    ImGui::Spacing();
    // Using the generic BeginCombo() API, you have full control over how to display the combo contents.
    // (your selection data could be an index, a pointer to the object, an id for the object, a flag intrusively
    // stored in the object itself, etc.)
    const char* combo_preview_value = filters[current_filter_dropdown_idx]->filter_name;  // Pass in the preview value visible before opening the combo (it could be anything)
    if (ImGui::BeginCombo("Select filter", combo_preview_value, flags)) {
        for (int n = 0; n < BASIC_FILTER_SIZE; n++) {
            const bool is_selected = (current_filter_dropdown_idx == n);
            if (ImGui::Selectable(filters[n]->filter_name, is_selected))
                current_filter_dropdown_idx = n;
                selected_filter = const_cast<filter*>(filters[current_filter_dropdown_idx]);
            // Set the initial focus when opening the combo (scrolling + keyboard navigation focus)
            if (is_selected)
                ImGui::SetItemDefaultFocus();
        }
        ImGui::EndCombo();
    }

    ImGui::Spacing();
    ImGui::Spacing();

    if(selected_filter->properties->lower_bound_strength == selected_filter->properties->upper_bound_strength) {
        // just print text instead of slider
        filter_strength = 0;
        ImGui::Text("Filter strength: %d (not adjustable)", filter_strength);
    }
    else {
        ImGui::SliderInt("Filter strength", &filter_strength, 
                    selected_filter->properties->lower_bound_strength, selected_filter->properties->upper_bound_strength, "%d", ImGuiSliderFlags_AlwaysClamp);
    }
    ImGui::Spacing();
    // check if selected filter is adjustable by size, if so then allow user to iterate through sizes
    if(selected_filter->properties->expandable_size) {
        int min_size = selected_filter->properties->sizes_avail[0];
        int max_size = selected_filter->properties->sizes_avail[selected_filter->properties->num_sizes_avail - 1];

        // Display a single slider for the range of values in sizes_avail
        ImGui::SliderInt("Filter size", &filter_size, min_size, max_size);

        // clamp filter_size such that it is odd i.e if user selects even then set it to odd
        if(filter_size % 2 == 0) {
            filter_size++;
            if(filter_size > max_size) {
                filter_size = max_size;
            }
            ImGui::Text("Filter size must be odd, setting to %d", filter_size);
        }
    }
    else {
        filter_size = 3;
        ImGui::Text("Filter size: %d (not adjustable)", filter_size);
    }
    ImGui::Spacing();
    ImGui::SliderInt("Filter passes (1 to 10)", &passes, 1, 10, "%d", ImGuiSliderFlags_AlwaysClamp);
    ImGui::Spacing();
    ImGui::SliderInt("Shift reds (-100 to 100%)", &red_strength, -100, 100, "%d%", ImGuiSliderFlags_AlwaysClamp);
    ImGui::Spacing();
    ImGui::SliderInt("Shift blues (-100 to 100%)", &blue_strength, -100, 100, "%d%", ImGuiSliderFlags_AlwaysClamp);
    ImGui::Spacing();
    ImGui::SliderInt("Shift greens (-100 to 100%)", &green_strength, -100, 100, "%d%", ImGuiSliderFlags_AlwaysClamp);
    ImGui::Spacing();
    ImGui::SliderInt("Shift alphas (-100 to 100%)", &alpha_strength, -100, 100, "%d%", ImGuiSliderFlags_AlwaysClamp);
    ImGui::Spacing();
    ImGui::SliderInt("Brightness (-100 to 100%)", &brightness, -100, 100, "%d%", ImGuiSliderFlags_AlwaysClamp);
    ImGui::Spacing();

    ImGui::Checkbox("Normalize image", &normalize);
    ImGui::Spacing();
    ImGui::Spacing();
    ImGui::Checkbox("Invert image", &invert);
    ImGui::Spacing();
    ImGui::Spacing();
    ImGui::Checkbox("Colour conversion", &conversion);
    ImGui::Spacing();
    ImGui::Spacing();
    ImGui::Checkbox("Colour threshold", &threshold);
    ImGui::Spacing();
    ImGui::Spacing();
    ImGui::Checkbox("Tint image", &show_tint);
    ImGui::Spacing();
    ImGui::Spacing();
    if(show_tint) {
        ImGui::Text("Tint colour of image");
        ImGui::Spacing();
        float w = (ImGui::GetContentRegionAvail().x - ImGui::GetStyle().ItemSpacing.y) * 0.40f;
        ImGui::SetNextItemWidth(w);
        ImGui::ColorPicker4("##tint1", (float*)&tint_colour, ImGuiColorEditFlags_AlphaBar |
            ImGuiColorEditFlags_PickerHueBar | ImGuiColorEditFlags_DisplayHex | ImGuiColorEditFlags_DisplayRGB | ImGuiColorEditFlags_DisplayHSV
            | ImGuiColorEditFlags_AlphaPreviewHalf | ImGuiColorEditFlags_HDR);
        
        ImGui::SameLine();
        ImGui::SetNextItemWidth(w);
        ImGui::ColorPicker4("##tint2", (float*)&tint_colour,  
            ImGuiColorEditFlags_PickerHueWheel | ImGuiColorEditFlags_NoInputs | ImGuiColorEditFlags_NoSidePreview);
        ImGui::Spacing();
        ImGui::Spacing();

        ImGui::SliderFloat("Tint strength (0 to 100%)", &blend_factor, 0.0f, 1.0f, "blend factor = %.3f", ImGuiSliderFlags_AlwaysClamp);

        ImGui::Spacing();
        ImGui::Spacing();
        ImGui::Spacing();
    }

    if(ImGui::Button("Apply changes")) {
        if(show_original) {
            show_preview = true;
            // pass kernel args to render function
            extra_args.red_shift = static_cast<char>(red_strength);
            extra_args.green_shift = static_cast<char>(green_strength);
            extra_args.blue_shift = static_cast<char>(blue_strength);
            extra_args.alpha_shift = static_cast<char>(alpha_strength);
            extra_args.brightness = static_cast<char>(brightness);

            extra_args.passes = static_cast<unsigned char>(passes);
            extra_args.normalize = normalize;
            extra_args.invert = invert;
            extra_args.conversion = conversion;
            extra_args.threshold = threshold;
            extra_args.filter_strength = static_cast<char>(filter_strength);
            extra_args.dimension = static_cast<unsigned char>(filter_size);
            extra_args.blend_factor = blend_factor;
            extra_args.tint[0] = tint_colour.x * 100;
            extra_args.tint[1] = tint_colour.y * 100;
            extra_args.tint[2] = tint_colour.z * 100;
            extra_args.tint[3] = tint_colour.w * 100;

            // time render applied changes and put it in console
            auto start = std::chrono::high_resolution_clock::now();
            ImGui::SetMouseCursor(ImGuiMouseCursor_NotAllowed);
            if(render_applied_changes(selected_filter->filter_name, extra_args, width, height, &texture_preview, channels,
                                    &image_data, &image_data_out, input, pixels_in, pixels_out)) {
                printf("Rendered changes successfully\n");
            }
            else {
                printf("Error rendering changes\n");
            }
            ImGui::SetMouseCursor(ImGuiMouseCursor_Arrow);
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> elapsed = end - start;
            printf("Time taken to render changes: %f seconds\n", elapsed.count());
            
            #ifdef _DEBUG
                printf("Using changes: \n");
                // print all values of extra args
                printf("Red shift: %d\n", extra_args.red_shift);
                printf("Green shift: %d\n", extra_args.green_shift);
                printf("Blue shift: %d\n", extra_args.blue_shift);
                printf("Alpha shift: %d\n", extra_args.alpha_shift);
                printf("Brightness: %d\n", extra_args.brightness);
                printf("Passes: %d\n", extra_args.passes);
                printf("Normalize: %d\n", extra_args.normalize);
                printf("Filter strength: %d\n", extra_args.filter_strength);
                printf("Filter size: %d\n", extra_args.dimension);
                printf("Blend factor: %f\n", extra_args.blend_factor);
                printf("Tint colour: %d, %d, %d, %d\n", extra_args.tint[0], extra_args.tint[1], extra_args.tint[2], extra_args.tint[3]);

                // print values of tint colour
                printf("Tint colour: %f, %f, %f, %f\n", tint_colour.x, tint_colour.y, tint_colour.z, tint_colour.w);
            #endif // DEBUG
        }
    }
    ImGui::SameLine();
    if(ImGui::Button("Clear all changes")) {
        show_preview = false;

        if(texture_preview != 0) {
            glDeleteTextures(1, &texture_preview);
            texture_preview = 0;
        }

    }
    ImGui::SameLine();
    if(ImGui::Button("Restore defaults")) {
        filter_strength = 0;
        red_strength = 0;
        green_strength = 0;
        blue_strength = 0;
        alpha_strength = 0;
        brightness = 0;
        normalize = false;
        blend_factor = 0;
        tint_colour = ImVec4(0.0f, 0.0f, 0.0f, 0.0f);
        show_tint = false;
    }

    ImGui::EndChild();

    ImGui::SetCursorPos(ImVec2(main_panel_size.x + 10, parent_cursor_start.y + side_panel_1_size.y));
    ImGui::BeginChild("Side panel 2", side_panel_2_size, true);
    ImGui::Text("Image details: ");
    ImGui::Spacing();
    if(show_original) {
        ImGui::Text("Width: %d", width);
        ImGui::Text("Height: %d", height);
        ImGui::Text("Channels: %d", channels);
        ImGui::Text("File size: %d bytes", width * height * channels);
        ImGui::Text("File path: %s", input);
        ImGui::Text("EXIF data: ");
        ImGui::Text("%s", exif_data_str.c_str());
        ImGui::Text("IPTC data: ");
        ImGui::Text("%s", iptc_data_str.c_str());
        ImGui::Text("XMP data: ");
        ImGui::Text("%s", xmp_data_str.c_str());
    }
    else {
        ImGui::Text("Please load an image to view details.");
    }
    ImGui::EndChild();
    ImGui::End();
}