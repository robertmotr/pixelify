#include <stdio.h>
#include <string>
#include <unistd.h>
#include "pixel.h"
#include "kernel.h"
#include <stdlib.h>
#include <assert.h>
#include "imgui.h"
#include "backends/imgui_impl_glfw.h"
#include "backends/imgui_impl_opengl3.h"
#define GL_SILENCE_DEPRECATION
#if defined(IMGUI_IMPL_OPENGL_ES2)
#include <GLES2/gl2.h>
#endif
#include <GLFW/glfw3.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

int8_t filter[] = {
  0, 0, 0,
  0, 1, 0,
  0, 0, 0
};

int8_t blur[] = {
  1, 1, 1,
  1, 1, 1,
  1, 1, 1
};

static void glfw_error_callback(int error, const char* description)
{
    fprintf(stderr, "GLFW Error %d: %s\n", error, description);
}

int main(int argc, char **argv) {
  if(argc != 3) {
    printf("Usage: %s <input> <output>\n", argv[0]);
    return 1;
  }

  std::string input = argv[1];
  std::string output = argv[2];

  glfwSetErrorCallback(glfw_error_callback);
    if (!glfwInit())
        return 1;

      // Decide GL+GLSL versions
  #if defined(IMGUI_IMPL_OPENGL_ES2)
      // GL ES 2.0 + GLSL 100
      const char* glsl_version = "#version 100";
      glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 2);
      glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
      glfwWindowHint(GLFW_CLIENT_API, GLFW_OPENGL_ES_API);
  #elif defined(__APPLE__)
      // GL 3.2 + GLSL 150
      const char* glsl_version = "#version 150";
      glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
      glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 2);
      glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);  // 3.2+ only
      glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);            // Required on Mac
  #else
      // GL 3.0 + GLSL 130
      const char* glsl_version = "#version 130";
      glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
      glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
      //glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);  // 3.2+ only
      //glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);            // 3.0+ only
  #endif

    // Create window with graphics context
    GLFWwindow* window = glfwCreateWindow(1280, 720, "Dear ImGui GLFW+OpenGL3 example", nullptr, nullptr);
    if (window == nullptr)
        return 1;
    glfwMakeContextCurrent(window);
    glfwSwapInterval(1); // Enable vsync

    // Setup Dear ImGui context
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;
    io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;     // Enable Keyboard Controls
    io.ConfigFlags |= ImGuiConfigFlags_NavEnableGamepad;      // Enable Gamepad Controls

    // Setup Dear ImGui style
    ImGui::StyleColorsDark();
    //ImGui::StyleColorsLight();

    // Setup Platform/Renderer backends
    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init(glsl_version);

    // Load Fonts
    // - If no fonts are loaded, dear imgui will use the default font. You can also load multiple fonts and use ImGui::PushFont()/PopFont() to select them.
    // - AddFontFromFileTTF() will return the ImFont* so you can store it if you need to select the font among multiple.
    // - If the file cannot be loaded, the function will return a nullptr. Please handle those errors in your application (e.g. use an assertion, or display an error and quit).
    // - The fonts will be rasterized at a given size (w/ oversampling) and stored into a texture when calling ImFontAtlas::Build()/GetTexDataAsXXXX(), which ImGui_ImplXXXX_NewFrame below will call.
    // - Use '#define IMGUI_ENABLE_FREETYPE' in your imconfig file to use Freetype for higher quality font rendering.
    // - Read 'docs/FONTS.md' for more instructions and details.
    // - Remember that in C/C++ if you want to include a backslash \ in a string literal you need to write a double backslash \\ !
    // - Our Emscripten build process allows embedding fonts to be accessible at runtime from the "fonts/" folder. See Makefile.emscripten for details.
    //io.Fonts->AddFontDefault();
    //io.Fonts->AddFontFromFileTTF("c:\\Windows\\Fonts\\segoeui.ttf", 18.0f);
    //io.Fonts->AddFontFromFileTTF("../../misc/fonts/DroidSans.ttf", 16.0f);
    //io.Fonts->AddFontFromFileTTF("../../misc/fonts/Roboto-Medium.ttf", 16.0f);
    //io.Fonts->AddFontFromFileTTF("../../misc/fonts/Cousine-Regular.ttf", 15.0f);
    //ImFont* font = io.Fonts->AddFontFromFileTTF("c:\\Windows\\Fonts\\ArialUni.ttf", 18.0f, nullptr, io.Fonts->GetGlyphRangesJapanese());
    //IM_ASSERT(font != nullptr);

    // Our state
    bool show_demo_window = true;
    bool show_another_window = false;
    ImVec4 clear_color = ImVec4(0.45f, 0.55f, 0.60f, 1.00f);

    while (!glfwWindowShouldClose(window)) {
        // Poll and handle events (inputs, window resize, etc.)
        // You can read the io.WantCaptureMouse, io.WantCaptureKeyboard flags to tell if dear imgui wants to use your inputs.
        // - When io.WantCaptureMouse is true, do not dispatch mouse input data to your main application, or clear/overwrite your copy of the mouse data.
        // - When io.WantCaptureKeyboard is true, do not dispatch keyboard input data to your main application, or clear/overwrite your copy of the keyboard data.
        // Generally you may always pass all inputs to dear imgui, and hide them from your application based on those two flags.
        glfwPollEvents();

        // Start the Dear ImGui frame
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        // 1. Show the big demo window (Most of the sample code is in ImGui::ShowDemoWindow()! You can browse its code to learn more about Dear ImGui!).
        if (show_demo_window)
            ImGui::ShowDemoWindow(&show_demo_window);

        // 2. Show a simple window that we create ourselves. We use a Begin/End pair to create a named window.
        {
            static float f = 0.0f;
            static int counter = 0;

            ImGui::Begin("Hello, world!");                          // Create a window called "Hello, world!" and append into it.

            ImGui::Text("This is some useful text.");               // Display some text (you can use a format strings too)
            ImGui::Checkbox("Demo Window", &show_demo_window);      // Edit bools storing our window open/close state
            ImGui::Checkbox("Another Window", &show_another_window);

            ImGui::SliderFloat("float", &f, 0.0f, 1.0f);            // Edit 1 float using a slider from 0.0f to 1.0f
            ImGui::ColorEdit3("clear color", (float*)&clear_color); // Edit 3 floats representing a color

            if (ImGui::Button("Button"))                            // Buttons return true when clicked (most widgets return true when edited/activated)
                counter++;
            ImGui::SameLine();
            ImGui::Text("counter = %d", counter);

            ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / io.Framerate, io.Framerate);
            ImGui::End();
        }

        // 3. Show another simple window.
        if (show_another_window)
        {
            ImGui::Begin("Another Window", &show_another_window);   // Pass a pointer to our bool variable (the window will have a closing button that will clear the bool when clicked)
            ImGui::Text("Hello from another window!");
            if (ImGui::Button("Close Me"))
                show_another_window = false;
            ImGui::End();
        }

        // Rendering
        ImGui::Render();
        int display_w, display_h;
        glfwGetFramebufferSize(window, &display_w, &display_h);
        glViewport(0, 0, display_w, display_h);
        glClearColor(clear_color.x * clear_color.w, clear_color.y * clear_color.w, clear_color.z * clear_color.w, clear_color.w);
        glClear(GL_COLOR_BUFFER_BIT);
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

        glfwSwapBuffers(window);
    }

    // Cleanup
    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();

    glfwDestroyWindow(window);
    glfwTerminate();

  int width, height, channels;
  // get image properties
  int ok = stbi_info(input.c_str(), &width, &height, &channels);
  if(ok != 1) {
    printf("Failed to get image properties: %s\n", stbi_failure_reason());
    return 1;
  }

  printf("Image properties:\n");
  printf("Width: %d\nHeight: %d\nChannels: %d\n", width, height, channels);
  // load image and get properties
  /*
    points to pixel data consists of *height scanlines of *width pixels,
    with each pixel consisting of N interleaved 8-bit components; the first
    pixel pointed to is top-left-most in the image. There is no padding between
    image scanlines or between pixels, regardless of format.
  */
  unsigned char* image_data = stbi_load(input.c_str(), &width, &height, &channels, 0);
  if (image_data == NULL) {
      printf("Failed to load image: %s\n", stbi_failure_reason());
      return 1;
  }

  unsigned char *image_output = new unsigned char[width * height * channels];

  if(channels == 3) {
    Pixel<3> *pixels_in = raw_image_to_pixel<3>(image_data, width * height);
    Pixel<3> *pixels_out = new Pixel<3>[width * height];

    run_kernel<3>(blur, 3, pixels_in, pixels_out, width, height);

    image_output = pixel_to_raw_image<3>(pixels_out, width * height);
  }
  else if(channels == 4) {
    Pixel<4> *pixels_in = raw_image_to_pixel<4>(image_data, width * height);
    Pixel<4> *pixels_out = new Pixel<4>[width * height];

    run_kernel<4>(blur, 3, pixels_in, pixels_out, width, height);

    image_output = pixel_to_raw_image<4>(pixels_out, width * height);
  }
  else {
    // not rgb/rgba so invalid 
    printf("Invalid # of channels.\n");
    return -1;
  }

  stbi_write_png(output.c_str(), width, height, channels, image_output, 0); 

  return 0;
}
